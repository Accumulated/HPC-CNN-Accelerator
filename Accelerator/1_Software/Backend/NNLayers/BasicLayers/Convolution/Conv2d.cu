#include "hip/hip_runtime.h"
#include "CommonInclude.h"
#include "Conv2d.h"


Conv2d:: Conv2d(SupportConvolutionOPs ConvType,
                int kernel_size,
                int stride,
                int padding,
                ActivationTypes activation_type,
                float* weight,
                float* bias):

            // Initialize the layer variables
            ConvType(ConvType),
            kernel_size(kernel_size),
            stride(stride),
            padding(padding),
            activation_type(activation_type){

    /* Missing output allocation and preperation */
    this -> weight = new Matrix();
    this -> bias = new Matrix();
}


Conv2d:: ~Conv2d() {

    /* */

}


Matrix* Conv2d::operator()(Matrix *D_input) {

   // The multiplication kernel is used for the 1x1 Conv2d and kxk Conv2d
    if (ConvType == CONV_1x1 || ConvType == CONV_KxK)
    {
        // Get number of blocks
        int nbx = (int) ceil((float)(this -> Output -> width) / (THREAD_GRANULARITY_BLOCKS * Tile_GEMM));
        int nby = (int) ceil((float)(this -> Output -> height) / Tile_GEMM);
        int num_block_for_phases = (int) ceil((float)(this -> weight -> width) / Tile_GEMM);

        // Check for zero blocks to make sure code runs correctly
        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        dim3 dim_Grid2(nbx, nby, 1);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        if (this -> bias -> elements != NULL)
        {
            this -> bias -> Matrix_SetDimensions(this -> Output -> height, 1, 1);

            // Call shared memory tiled Multiplication  algorithm
            MatrixMulKernel<<<dim_Grid2, dim_Block2>>> (

                this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                D_input -> elements, D_input -> height, D_input -> width, D_input -> depth,

                this -> Output -> elements, this -> Output -> height, this -> Output -> width, this -> Output -> depth,

                num_block_for_phases, activation_type,

                BIASED, this -> bias -> elements

            );
        }
        else
        {
            // Call shared memory tiled Multiplication  algorithm
            MatrixMulKernel<<<dim_Grid2, dim_Block2>>> (

                this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                D_input -> elements, D_input -> height, D_input -> width, D_input -> depth,

                this -> Output -> elements, this -> Output -> height, this -> Output -> width, this -> Output -> depth,

                num_block_for_phases, this -> activation_type,

                NOT_BIASED, NULL

            );
        }
    }

    // This case is for DWConv2d
    else if (ConvType == CONV_DW)
    {

        int nbx = (int)ceil((float)(this -> Output -> width) / Tile_GEMM);
        int nby = (int)ceil((float)(this -> Output -> height) / Tile_GEMM);
        int nbz = this -> Output -> depth;

        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        // This is the only kernel that runs 3d Grid;
        // Each block in z dimension controls 1 channel
        dim3 dim_Grid2(nbx, nby, nbz);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        DWConv2d_kernel<<<dim_Grid2, dim_Block2>>> (

            D_input -> elements, D_input -> height, D_input -> width, D_input -> depth,

            this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

            this -> Output -> elements, this -> Output -> height, this -> Output -> width, this -> Output -> depth,

            this -> stride
        );

    }

    else{

        std::cout << "Unsupported Convolution Operation" << std::endl;

    }

    return this -> Output;
}
