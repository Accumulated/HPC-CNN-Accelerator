#include "hip/hip_runtime.h"
#include "CommonInclude.h"
#include "Matrix.h"
#include "kernels.h"
#include "Conv2d.h"


Conv2d:: Conv2d(SupportConvolutionOPs ConvType,
                int kernel_size,
                int stride,
                int padding,
                ActivationTypes activation_type,
                Matrix* weight,
                Matrix* bias,
                Matrix* input,
                Matrix* output):

            // Initialize the layer variables
            ConvType(ConvType),
            kernel_size(kernel_size),
            stride(stride),
            padding(padding),
            activation_type(activation_type),
            weight(weight),
            bias(bias),
            input(input),
            output(output) {


}


Conv2d:: ~Conv2d() {

    /* */

}


void Conv2d::operator()() {

   // The multiplication kernel is used for the 1x1 Conv2d and kxk Conv2d
    if (ConvType == CONV_1x1 || ConvType == CONV_KxK)
    {
        // Get number of blocks
        int nbx = (int) ceil((float)(this -> output -> width) / (THREAD_GRANULARITY_BLOCKS * Tile_GEMM));
        int nby = (int) ceil((float)(this -> output -> height) / Tile_GEMM);
        int num_block_for_phases = (int) ceil((float)(this -> weight -> width) / Tile_GEMM);

        // Check for zero blocks to make sure code runs correctly
        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        dim3 dim_Grid2(nbx, nby, 1);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        if (this -> bias -> elements != NULL)
        {
            this -> bias -> Matrix_SetDimensions(this -> output -> height, 1, 1);

            // Call shared memory tiled Multiplication  algorithm
            MatrixMulKernel<<<dim_Grid2, dim_Block2>>> (

                this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                this -> input -> elements, this -> input -> height, this -> input -> width, this -> input -> depth,

                this -> output -> elements, this -> output -> height, this -> output -> width, this -> output -> depth,

                num_block_for_phases, activation_type,

                BIASED, this -> bias -> elements

            );
        }
        else
        {
            // Call shared memory tiled Multiplication  algorithm
            MatrixMulKernel<<<dim_Grid2, dim_Block2>>> (

                this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                this -> input -> elements, this -> input -> height, this -> input -> width, this -> input -> depth,

                this -> output -> elements, this -> output -> height, this -> output -> width, this -> output -> depth,

                num_block_for_phases, this -> activation_type,

                NOT_BIASED, NULL

            );
        }
    }

    // This case is for DWConv2d
    else if (ConvType == CONV_DW)
    {

        int nbx = (int)ceil((float)(this -> output -> width) / Tile_GEMM);
        int nby = (int)ceil((float)(this -> output -> height) / Tile_GEMM);
        int nbz = this -> output -> depth;

        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        // This is the only kernel that runs 3d Grid;
        // Each block in z dimension controls 1 channel
        dim3 dim_Grid2(nbx, nby, nbz);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        DWConv2d_kernel<<<dim_Grid2, dim_Block2>>> (

            this -> input -> elements, this -> input -> height, this -> input -> width, this -> input -> depth,

            this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

            this -> output -> elements, this -> output -> height, this -> output -> width, this -> output -> depth,

            this -> stride
        );

    }

    else{

        std::cout << "Unsupported Convolution Operation" << std::endl;

    }

}
