#include "hip/hip_runtime.h"
#include "CommonInclude.h"
#include "BatchNorm.h"


BatchNorm:: BatchNorm(const BatchNorm_Weights* Details, ActivationTypes activation, Dimension* InputDim){


  if(Details -> Mean && Details -> Variance && Details -> Weights && Details -> Bias){

    /* Missing output allocation and preperation */
    this -> mean = new Matrix(sizeof(Details -> Mean) / sizeof(float),
                              1,
                              1,
                              Details -> Mean,
                              DefineOnDevice);

    this -> bias = new Matrix(sizeof(Details -> Bias) / sizeof(float),
                              1,
                              1,
                              Details -> Bias,
                              DefineOnDevice);

  this -> weights = new Matrix(sizeof(Details -> Weights) / sizeof(float),
                              1,
                              1,
                              Details -> Weights,
                              DefineOnDevice);

  this -> variance = new Matrix(sizeof(Details -> Variance) / sizeof(float),
                                1,
                                1,
                                Details -> Variance,
                                DefineOnDevice);
  }

  this -> activation = activation;

  this -> OutputDim = Dimension{
                        .Height = InputDim -> Height,
                        .Width = InputDim -> Width,
                        .Depth = InputDim -> Depth
                      };

  this -> Output = new Matrix(this -> OutputDim.Height,
                              this -> OutputDim.Width,
                              this -> OutputDim.Depth,
                              NULL,
                              DefineOnDevice);

}


Matrix* BatchNorm:: operator()(Matrix *D_input)
{

    /* The D_input matrix is a device matrix */

    /*
      All weights, bias, running mean and running variance
      are pre-defined. Just call the function and use the
      matrices.

      All bias, weights, mean and bariance matrices are 1x1xC

      Output Matrix is modified by the equation
      (y = ((x - Mean) / (sqrt(variance) + epsilon)) * weights + bais)
    */

    int nbx = (int)ceil((float)D_input -> width / Tile_GEMM);
    int nby = (int)ceil((float)D_input -> height / Tile_GEMM);
    int nbz = D_input -> depth;

    if (nbx == 0) nbx = 1;
    if (nby == 0) nby = 1;

    // This is the only kernel that runs 3d Grid;
    // Each block in z dimension controls 1 channel
    dim3 dim_Grid3(nbx, nby, nbz);
    dim3 dim_Block3(Tile_GEMM, Tile_GEMM, 1);

    BatchNormKernel <<< dim_Grid3, dim_Block3 >>> (

      D_input -> elements,

      this -> Output -> elements,
      this -> Output -> height,
      this -> Output -> width,
      this -> Output -> depth,

      this -> mean -> elements, this -> variance -> elements,
      this -> weights -> elements, this -> bias -> elements,

      this -> activation

    );


    return this -> Output;
}
