#include "hip/hip_runtime.h"
#include "CommonInclude.h"
#include "Conv2d.h"

using namespace std;

Conv2d:: Conv2d(SupportConvolutionOPs ConvType,
                int stride,
                int padding,
                ActivationTypes activation_type,
                const ConvDetails * Details,
                Dimension* InputDim):

            // Initialize the layer variables
            ConvType(ConvType),
            stride(stride),
            padding(padding),
            activation_type(activation_type),
            InputDim(InputDim){

    // Allocate an array of Matrix pointers
    this -> Output = new Matrix*[this -> numberOfStreams];

    if(Details -> ConvWeights){

        /* Missing output allocation and preperation */
        this -> weight = new Matrix(Details -> FilterHeight,
                                    Details -> FilterWidth,
                                    Details -> FilterDepth,
                                    Details -> FilterDensity,
                                    Details -> ConvWeights,
                                    DefineOnDevice);
    }

    if(Details -> Bias){

        this -> bias = new Matrix(Details -> FilterDensity,
                                    1,
                                    1,
                                    Details -> Bias,
                                    DefineOnDevice);
    }

    // Height and width changes, Only depth remains still
    int OutputHeight = (InputDim -> Height + 2 * padding - Details -> FilterHeight)/stride + 1;
    int OutputWidth = (InputDim -> Width + 2 * padding - Details -> FilterWidth)/stride + 1;
    int OutputDepth = 0;

    if(ConvType == CONV_1x1){

        // Output depth is the number of filters available (Density)
        OutputDepth = Details -> FilterDensity;

        this -> OutputDim = Dimension{
            .Height = OutputHeight,
            .Width = OutputWidth,
            .Depth = OutputDepth
        };

        // Modify Filter Matrix to have dimensions ((K^2 * M) x C x 1); K = 1
        this -> weight -> Matrix_SetDimensions(
            Details -> FilterHeight * Details -> FilterWidth * Details -> FilterDensity,
            Details -> FilterDepth,
            1
        );


        for (int i = 0; i < this -> numberOfStreams; i++) {

            // Modify Output Matrix preprocessing to have dimesions ((K^2 * M) x (H * W) x 1);
            this -> Output[i] = new Matrix(
                                // (K^2 * M)
                                Details -> FilterHeight * Details -> FilterWidth * Details -> FilterDensity,
                                // (H * W)
                                OutputHeight * OutputWidth,
                                // 1 channel
                                1,
                                NULL,
                                DefineOnDevice
                            );

        }

    }
    else if(ConvType == CONV_KxK){

        // Output depth is the number of filters available (Density)
        OutputDepth = Details -> FilterDensity;


        this -> OutputDim = Dimension{
            .Height = OutputHeight,
            .Width = OutputWidth,
            .Depth = OutputDepth
        };


        this -> Conv_InputUnrolling = new InputUnrolling(
                                            InputDim,
                                            Details,
                                            padding,
                                            stride
                                        );

        // Modify Output Matrix preprocessing to have dimesions ((K^2 * M) x (H * W) x 1);
        // Convolution output has dimensions of (M x (H_out * W_out) x 1)
        for (int i = 0; i < this -> numberOfStreams; i++) {
            this -> Output[i] = new Matrix(
                                    // (K^2 * M)
                                    Details -> FilterDensity,
                                    // (H * W)
                                    OutputHeight * OutputWidth,
                                    // 1 channel
                                    1,
                                    NULL,
                                    DefineOnDevice
                                );
        }

    }
    else if(ConvType == CONV_DW){

        // Output depth is the same as the input depth
        OutputDepth = InputDim -> Depth;

        this -> OutputDim = Dimension{
            .Height = OutputHeight,
            .Width = OutputWidth,
            .Depth = OutputDepth
        };

        for (int i = 0; i < this -> numberOfStreams; i++) {
            this -> Output[i] = new Matrix(
                                OutputHeight,
                                OutputWidth,
                                OutputDepth,
                                NULL,
                                DefineOnDevice
                            );
        }
    }

    else{

        std::cout << "Unsupported Convolution Operation" << std::endl;

    }

    if(padding){

        this -> pad = new PaddingLayer(InputDim, this -> padding);

    }

}


Dimension* Conv2d:: Conv2d_GetOutputDim() {

    return &(this -> OutputDim);

}


Conv2d:: ~Conv2d() {

    /* */

}


Matrix** Conv2d::operator()(Matrix **D_input) {

    /* First: Some preprocessing. */


   // The multiplication kernel is used for the 1x1 Conv2d and kxk Conv2d
    if (ConvType == CONV_1x1)
    {

        /*
            CONV_1x1
            // Modify Input matrix to have dimensions (C x (H * W) x 1)
            Set_HostMatrix(InputIMG -> depth, InputIMG -> height * InputIMG -> width, 1, InputIMG);
        */

        for (int i = 0; i < this -> numberOfStreams; i++) {
            // Modify Input matrix to have dimensions (C x (H * W) x 1)
            D_input[i]->Matrix_SetDimensions(
                D_input[i] -> depth,
                D_input[i] -> height * D_input[i] -> width,
                1
            );
        }

        // Get number of blocks
        int nbx = (int) ceil((float)(this -> Output[0] -> width) / (THREAD_GRANULARITY_BLOCKS * Tile_GEMM));
        int nby = (int) ceil((float)(this -> Output[0] -> height) / Tile_GEMM);
        int num_block_for_phases = (int) ceil((float)(this -> weight -> width) / Tile_GEMM);

        // Check for zero blocks to make sure code runs correctly
        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        dim3 dim_Grid2(nbx, nby, 1);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        if (this -> bias != NULL)
        {

            for (int i = 0; i < this -> numberOfStreams; i++) {

                this -> bias -> Matrix_SetDimensions(this -> Output[i] -> height, 1, 1);

                // Call shared memory tiled Multiplication  algorithm
                MatrixMulKernel<<<dim_Grid2, dim_Block2, 0, this -> streams[i]>>> (

                    this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                    D_input[i] -> elements, D_input[i] -> height, D_input[i] -> width, D_input[i] -> depth,

                    this -> Output[i] -> elements, this -> Output[i] -> height, this -> Output[i] -> width, this -> Output[i] -> depth,

                    num_block_for_phases, activation_type,

                    BIASED, this -> bias -> elements

                );
            }
        }
        else
        {
            for (int i = 0; i < this -> numberOfStreams; i++) {

                // Call shared memory tiled Multiplication  algorithm
                MatrixMulKernel<<<dim_Grid2, dim_Block2, 0, this -> streams[i]>>> (

                    this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                    D_input[i] -> elements, D_input[i] -> height, D_input[i] -> width, D_input[i] -> depth,

                    this -> Output[i] -> elements, this -> Output[i] -> height, this -> Output[i] -> width, this -> Output[i] -> depth,

                    num_block_for_phases, this -> activation_type,

                    NOT_BIASED, NULL

                );
            }
        }
    }


    else if(ConvType == CONV_KxK){

        Matrix **ptr = D_input;

        if(this -> padding){
            ptr = (*this -> pad)(D_input);
        }

        // 1st phase: Filter unrolling and Input unrolling
        Matrix **FilterUnrolled = (*this -> Conv_InputUnrolling)();
        Matrix **InputUnrolled = (*this -> Conv_InputUnrolling)(ptr);

        // Get number of blocks
        int nbx = (int) ceil((float)(this -> Output[0] -> width) / (THREAD_GRANULARITY_BLOCKS * Tile_GEMM));
        int nby = (int) ceil((float)(this -> Output[0] -> height) / Tile_GEMM);
        int num_block_for_phases = (int) ceil((float)(FilterUnrolled[0] -> width) / Tile_GEMM);

        // Check for zero blocks to make sure code runs correctly
        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        dim3 dim_Grid2(nbx, nby, 1);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        if (this -> bias != NULL)
        {

            for (int i = 0; i < this -> numberOfStreams; i++) {

                this -> bias -> Matrix_SetDimensions(this -> Output[i] -> height, 1, 1);

                // Call shared memory tiled Multiplication  algorithm
                MatrixMulKernel<<<dim_Grid2, dim_Block2, 0, this -> streams[i]>>> (

                    FilterUnrolled[i] -> elements, FilterUnrolled[i] -> height, FilterUnrolled[i] -> width, FilterUnrolled[i] -> depth,

                    InputUnrolled[i] -> elements, InputUnrolled[i] -> height, InputUnrolled[i] -> width, InputUnrolled[i] -> depth,

                    this -> Output[i] -> elements, this -> Output[i] -> height, this -> Output[i] -> width, this -> Output[i] -> depth,

                    num_block_for_phases, activation_type,

                    BIASED, this -> bias -> elements

                );
            }
        }
        else
        {
            for (int i = 0; i < this -> numberOfStreams; i++) {

                // Call shared memory tiled Multiplication  algorithm
                MatrixMulKernel<<<dim_Grid2, dim_Block2, 0, this -> streams[i]>>> (

                    FilterUnrolled[i] -> elements, FilterUnrolled[i] -> height, FilterUnrolled[i] -> width, FilterUnrolled[i] -> depth,

                    InputUnrolled[i] -> elements, InputUnrolled[i] -> height, InputUnrolled[i] -> width, InputUnrolled[i] -> depth,

                    this -> Output[i] -> elements, this -> Output[i] -> height, this -> Output[i] -> width, this -> Output[i] -> depth,

                    num_block_for_phases, this -> activation_type,

                    NOT_BIASED, NULL

                );

            }
        }
    }


    // This case is for DWConv2d
    else if (ConvType == CONV_DW)
    {

        Matrix **ptr = D_input;

        if(this -> padding){
            ptr = (*this -> pad)(D_input);
        }

        int nbx = (int)ceil((float)(this -> Output[0] -> width) / Tile_GEMM);
        int nby = (int)ceil((float)(this -> Output[0] -> height) / Tile_GEMM);
        int nbz = this -> Output[0] -> depth;

        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        // This is the only kernel that runs 3d Grid;
        // Each block in z dimension controls 1 channel
        dim3 dim_Grid2(nbx, nby, nbz);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);

        for (int i = 0; i < this -> numberOfStreams; i++) {

            DWConv2d_kernel<<<dim_Grid2, dim_Block2, 0, this -> streams[i]>>> (

                ptr[i] -> elements, ptr[i] -> height, ptr[i] -> width, ptr[i] -> depth,

                this -> weight -> elements, this -> weight -> height, this -> weight -> width, this -> weight -> depth,

                this -> Output[i] -> elements, this -> Output[i] -> height, this -> Output[i] -> width, this -> Output[i] -> depth,

                this -> stride
            );
        }

    }

    else{

        std::cout << "Unsupported Convolution Operation" << std::endl;

    }

    for (int i = 0; i < this -> numberOfStreams; i++) {

        this -> Output[i] -> height = this -> OutputDim.Height;
        this -> Output[i] -> width = this -> OutputDim.Width;
        this -> Output[i] -> depth = this -> OutputDim.Depth;

    }

    return this -> Output;
}
